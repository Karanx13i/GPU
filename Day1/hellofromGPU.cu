
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void helloFromGPU() {
    printf("Hello from GPU thread!\n");
}

int main() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

